
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

#include <time.h>

#include <iostream>

#include "location.cu"	
#include "particle.cu"

#define N 50

using namespace std;

__device__ Particle dev_particles[N];
Particle particles[N];

__device__ Position dev_globalOptimum;

void initParticles() {
	srand(time(NULL));
	for (size_t i = 0; i < N; i++)
	{
		float x = rand();
		float y = rand();
		Position location = Position(&x, &y);
		particles[i].position = &location;

	}
} 
int main()
{
	//initialize particles with random positions (on host)
	initParticles();

	//copy particles from host to device
	hipMemcpyToSymbol(HIP_SYMBOL(particles), dev_particles, N * sizeof(Particle));

    return 0;
}

void checkError() {
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
	}
}
