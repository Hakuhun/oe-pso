#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include "vector_functions.h"
#include <hiprand/hiprand_kernel.h>
#include <ctime>

#include <stdio.h>
#include <stdlib.h>

#include <time.h>

#include <iostream>

#define N 1000
#define MIN2 0
#define MAX2 50
#define RANDOM(a, b) rand()%(MAX2-MIN2+1)+(MIN2)
#define fitness(x) DistanceCalculate()

using namespace std;

class Managed {
public:
	void *operator new(size_t len) {
		void *ptr;
		hipMallocManaged(&ptr, len);
		hipDeviceSynchronize();
		return ptr;
	}

	void operator delete(void *ptr) {
		hipDeviceSynchronize();
		hipFree(ptr);
	}
};

//class Location : public Managed
//{
//public:
//	float x;
//	float y;
//
//	Location() {}
//
//	Location(float x, float y) {
//		this->x = x;
//		this->y = y;
//	}
//
//	Location operator=(const Location &mng) {
//		x = mng.x;
//		y = mng.y;
//		return *this;
//	}
//
//	//K�t vektor k�l�nbs�ge
//	Location operator-(const Location &mng) {
//		return Location(this->x - mng.x, this->y - mng.y);
//	}
//
//	//K�t vektor �sszege
//	Location operator+(const Location * mng) {
//		Location result = Location(this->x + mng->x, this->y + mng->y);
//		return result;
//	}
//
//	//sz�mmal val� szorz�s
//	Location operator*(const int &number) {
//		return Location(this->x * number, this->y * number);
//	}
//};

class Particle : public Managed
{
public:

	Particle() {

	}

	//vector of the current location
	float2 position;
	//vector of the particle's local optimum
	float2 localOptimum;
	//vector of the position where the particle is heading to
	float2 direction;

	float2 velocity = make_float2(0, 0);
};

__device__ Particle * dev_particles[N];
Particle host_particles[N];

__shared__ float2 dev_globalOptimum;

//Innertial coefficent (innerci�lis egy�tthat�)
__device__ float w = 0.5;

//Acceleration coefficent (gyors�t�si egy�tthat�)
__device__ float c1 = 0.2;

//Acceleration coefficent (gyors�t�si egy�tthat�)
__device__ float c2 = 0.2;

inline __host__ __device__ float2 operator-(float2 a, float2 b)
{
	return make_float2(a.x - b.x, a.y - b.y);
}

inline __host__ __device__ float2 operator+(float2 a, float2 b)
{
	return make_float2(a.x + b.x, a.y + b.y);
}

inline __host__ __device__ float2 operator*(float2 a, float2 b)
{
	return make_float2(a.x * b.x, a.y * b.y);
}

inline __host__ __device__ float2 operator*(float2 a, int b)
{
	return make_float2(a.x * b, a.y * b);
}

inline __host__ __device__ float2 operator*(int b, float2 a)
{
	return make_float2(a.x * b, a.y * b);
}

//fitness 
__device__ double DistanceCalculate(float2 a, float2 b)
{
	float2 diff = a - b;
	return sqrt(pow(diff.x, 2) + pow(diff.y, 2));
}

__global__ void Evaluation() {

	int index = blockDim.x * blockIdx.x + threadIdx.x;
	Particle * particle = dev_particles[index];

	if (DistanceCalculate(particle->direction, particle->localOptimum) <
		DistanceCalculate(particle->direction - particle->velocity, particle->localOptimum))
	{
		particle->localOptimum = particle->direction;

		if (DistanceCalculate(particle->direction, particle->localOptimum) <
			DistanceCalculate(particle->direction - particle->velocity, dev_globalOptimum))
		{
			dev_globalOptimum = particle->direction;
		}
	}
}

__device__ float cudaRand()
{
	int tId = threadIdx.x + (blockIdx.x * blockDim.x);
	hiprandState state;
	hiprand_init((unsigned long long)clock() + tId, 0, 0, &state);

	return hiprand_uniform_double(&state);
}

__global__ void CalculateVelocity() {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	Particle * particle = dev_particles[index];

	//Calculate the velocity
	particle->velocity = w * particle->velocity
		+ cudaRand() * c1 * (particle->localOptimum - particle->direction)
		+ cudaRand() * c2 * (dev_globalOptimum - particle->direction);
}

__global__ void CalculateNewDirection() {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	Particle * particle = dev_particles[index];

	particle->direction = particle->direction + particle->velocity;
}

void initParticles() {
	for (size_t i = 0; i < N; i++)
	{
		srand(time(NULL));
		host_particles[i] = Particle();
		host_particles[i].position = make_float2(RANDOM(MIN2, MAX2), RANDOM(MIN2, MAX2));
		host_particles[i].localOptimum = make_float2(RANDOM(MIN2, MAX2), RANDOM(MIN2, MAX2));
		host_particles[i].direction = make_float2(RANDOM(MIN2, MAX2), RANDOM(MIN2, MAX2));
	}
}

void checkError() {
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		cout << stderr, hipGetErrorName(cudaStatus);
	}
}

int main()
{
	//initialize particles with random positions (on host)
	initParticles();

	//copy particles from host to device
	hipMemcpyToSymbol(HIP_SYMBOL(dev_particles), host_particles, N * sizeof(Particle));
	checkError();
	//initalize global optimum variable
	float2 host_gOptimum = make_float2(1, 1);
	hipMemcpyToSymbol(HIP_SYMBOL(&dev_globalOptimum), &host_gOptimum, N * sizeof(float2));
	checkError();

	Evaluation << <1, N >> > ();
	checkError();

	int i = 0;
	
	while (i < 1000)
	{
		CalculateVelocity << <1, N >> > ();
		checkError();
		CalculateNewDirection << <1, N >> > ();
		checkError();
		Evaluation << <1, N >> > ();
		checkError();
		i++;
	}
	cout << "Vege";
	cin.get();

    return 0;
}