#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"
#include "vector_functions.h"
#include <hiprand/hiprand_kernel.h>
#include <ctime>

#include <stdio.h>
#include <stdlib.h>

#include <time.h>

#include <iostream>

#define N 100
#define MIN2 0
#define MAX2 50
#define RANDOM(a, b) rand()%(a-b+1)+(a)

using namespace std;

typedef struct PSOParticle {
	//vector of the current location
	float2 position;
	//vector of the particle's local optimum
	float2 localOptimum;
	//vector of the position where the particle is heading to
	float2 direction;

	float2 velocity;
}Particle;

__device__ Particle dev_particles[N];
Particle host_particles[N];

__device__ float2 dev_globalOptimum;

//Innertial coefficent (innerci�lis egy�tthat�)
__constant__ float w = 0.5;

//Acceleration coefficent (gyors�t�si egy�tthat�)
__constant__ float c1 = 0.2;

//Acceleration coefficent (gyors�t�si egy�tthat�)
__constant__ float c2 = 0.2;

inline __host__ __device__ float2 operator-(float2 a, float2 b)
{
	return make_float2(a.x - b.x, a.y - b.y);
}

inline __host__ __device__ float2 operator+(float2 a, float2 b)
{
	return make_float2(a.x + b.x, a.y + b.y);
}

inline __host__ __device__ float2 operator*(float2 a, float2 b)
{
	return make_float2(a.x * b.x, a.y * b.y);
}

inline __host__ __device__ float2 operator*(float2 a, int b)
{
	return make_float2(a.x * b, a.y * b);
}

inline __host__ __device__ float2 operator*(int b, float2 a)
{
	return make_float2(a.x * b, a.y * b);
}

//fitness 
__device__ double DistanceCalculate(float2 a, float2 b)
{
	float2 diff = a - b;
	return sqrt(pow(diff.x, 2) + pow(diff.y, 2));
}

__global__ void Evaluation() {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	Particle * particle = &dev_particles[index];

	if (DistanceCalculate(particle->direction, particle->localOptimum) <
		DistanceCalculate(particle->direction - particle->velocity, particle->localOptimum))
	{
		particle->localOptimum = particle->direction;

		if (DistanceCalculate(particle->direction, particle->localOptimum) <
			DistanceCalculate(particle->direction - particle->velocity, dev_globalOptimum))
		{
			dev_globalOptimum = particle->direction;
		}
	}
	__syncthreads();
}

__device__ float cudaRand()
{
	int tId = threadIdx.x + (blockIdx.x * blockDim.x);
	hiprandState state;
	hiprand_init((unsigned long long)clock() + tId, 0, 0, &state);

	return hiprand_uniform_double(&state);
}

__device__ float cudaRandRange(int min, int max)
{
	int tId = threadIdx.x + (blockIdx.x * blockDim.x);
	hiprandState state;
	hiprand_init((unsigned long long)clock() + tId, 0, 0, &state);
	float myrandf = hiprand_uniform_double(&state);
	myrandf *= (max-min + 0.999999);
	return myrandf;
}

__global__ void CalculateVelocity() {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	Particle * particle = &dev_particles[index];

	//Calculate the velocity
	particle->velocity = w * particle->velocity
		+ cudaRand() * c1 * (particle->localOptimum - particle->direction)
		+ cudaRand() * c2 * (dev_globalOptimum - particle->direction);
}

__global__ void CalculateNewDirection() {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	Particle * particle = &dev_particles[index];

	particle->direction = particle->direction + particle->velocity;
	__syncthreads();
}

__global__ void checkParticles(){
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	Particle particle = dev_particles[index];
	printf("(%d) x: %.2f, y : %.2f\n",index, particle.direction.x, particle.direction.y);
	__syncthreads();
}

void initParticles() {
	srand(time(NULL));
	for (size_t i = 0; i < N; i++)
	{
		host_particles[i] = Particle();
		host_particles[i].position = make_float2(RANDOM(MIN2, MAX2), RANDOM(MIN2, MAX2));
		host_particles[i].localOptimum = make_float2(RANDOM(MIN2, MAX2), RANDOM(MIN2, MAX2));
		host_particles[i].direction = make_float2(RANDOM(MIN2, MAX2), RANDOM(MIN2, MAX2));
	}
}

void checkError() {
	hipError_t cudaStatus = hipGetLastError();	
	if (cudaStatus != hipSuccess) {
		cout << "Hiba: " << hipGetErrorName(cudaStatus) << '\n';
		cout << hipGetErrorString(cudaStatus);
	}
}

int main()
{
	//initialize particles with random positions (on host)
	initParticles();

	//copy particles from host to device
	hipMemcpyToSymbol(HIP_SYMBOL(dev_particles), host_particles, N * sizeof(Particle));
	checkError();

	//initalize global optimum variable
	float2 host_gOptimum = make_float2(1, 1);
	hipMemcpyToSymbol(HIP_SYMBOL(dev_globalOptimum), &host_gOptimum.y, sizeof(float2));
	checkError();

	checkParticles << <1, N >> > ();

	Evaluation << <1, N >> > ();
	checkError();

	int i = 0;
	
	while (i < 100)
	{
		CalculateVelocity << <1, N >> > ();
		checkError();
		CalculateNewDirection << <1, N >> > ();
		checkError();
		Evaluation << <1, N >> > ();
		checkError();
		checkParticles << <1, N >> > ();
		//system("cls");
		i++;
	}
	//cout << "Vege";
	cin.get();

    return 0;
}